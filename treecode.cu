#include "hip/hip_runtime.h"
#include "treedefs.h"
#include "CudaArrayCopyUtils.h"
#include "treecodeCU.h"
#include "cudahelper.h"
#include <iostream>

//: We should really be using native CUDA vectors for this.... but that requires more funny typing magic to convert the CPU data

template<size_t DIM, typename Float, size_t PPG> __device__ bool passesMAC(const GroupInfo<DIM, Float, PPG>& groupInfo, const Node<DIM, Float>& nodeHere, Float theta) {
	
	Float d = mag(groupInfo.center - nodeHere.barycenter.pos) - groupInfo.radius;
	Float l = 2 * nodeHere.radius;
	return d > (l / theta);
	
}


template<template<size_t, typename> class ElemType, template<size_t, typename> class ElemTypeArray, size_t DIM, typename Float>
__device__ void initStack(ElemTypeArray<DIM, Float> level, size_t levelCt, ElemTypeArray<DIM, Float> stack, size_t* stackCt){
	if(threadIdx.x < levelCt){
		if(levelCt > level.elems || threadIdx.x >= stack.elems) printf("%d.%d %s: %lu (should be <= %lu) elements. stackCt @ %d / %lu\n",blockIdx.x, threadIdx.x, __func__,levelCt,level.elems,threadIdx.x,stack.elems);

		ElemType<DIM, Float> eHere;
		level.get(threadIdx.x, eHere);
		stack.set(threadIdx.x, eHere);
	}
	if(threadIdx.x == 0){
		atomicExch((unsigned long long*)stackCt,(unsigned long long)levelCt); // Don't want to have to threadfence afterwards. Just make sure it's set!
	}
}

template<size_t DIM, typename Float> __device__ void dumpStackChildren(NodeArray<DIM, Float> stack, const size_t* stackCt){
	size_t dst = *stackCt;
	for(size_t i = 0; i < dst; i++){
			printf("(%lu) see (%lu %lu) in (%p/%lu)\n",i,stack.childStart[i],stack.childCount[i], stackCt, dst);
	}
}

template<template<size_t, typename> class ElemType, template<size_t, typename> class ElemTypeArray, size_t DIM, typename Float> __device__ void pushAll(const ElemTypeArray<DIM, Float> src, const size_t srcCt, ElemTypeArray<DIM, Float> stack, size_t* stackCt){
	// This is a weird compiler bug. There's no reason this shouldn't have worked without the cast.
	size_t dst = atomicAdd((unsigned long long*)stackCt, (unsigned long long)srcCt);
	if(srcCt > src.elems || dst >= stack.elems)  printf("%d.%d %s: %lu (should be <= %lu) elements. stackCt @ %lu / %lu\n",blockIdx.x, threadIdx.x, __func__, srcCt,src.elems,dst,stack.elems);
	for(size_t i = dst, j = 0; i < dst + srcCt; i++, j++){
		ElemType<DIM, Float> eHere;
		src.get(j, eHere);
		stack.set(i, eHere);
	}
}

// Needs softening
template<size_t DIM, typename Float> __device__ Vec<DIM, Float> calc_force(const PointMass<DIM, Float> &m1, const PointMass<DIM, Float> &m2, Float softening){
	Vec<DIM, Float> disp = m1.pos - m2.pos;
	Vec<DIM, Float> force;
	//Another CUDA-induced casting hack here. Otherwise it tries to call the device version of the code
	force = disp * ((m1.m * m2.m) / (Float)(softening + pow((Float)mag_sq(disp),(Float)1.5)));
	return force;
}

template<size_t DIM, typename Float, TraverseMode Mode> __device__ InteractionType<DIM, Float, Mode> freshInteraction(){
	InteractionType<DIM, Float, Mode> fresh; for(size_t i = 0; i < DIM; i++){
		fresh.x[i] = 0.0;
	}
	return fresh;
}

template<typename T> __device__ inline void swap(T& a, T& b){
	T c(a); a=b; b=c;
}

template<size_t DIM, typename Float, size_t PPG, size_t MAX_LEVELS, size_t INTERACTION_THRESHOLD, TraverseMode Mode>
__global__ void traverseTreeKernel(const size_t nGroups, const GroupInfoArray<DIM, Float, PPG> groupInfo,
								   const size_t startDepth, NodeArray<DIM, Float>* treeLevels, const size_t* treeCounts,
								   const size_t n, const ParticleArray<DIM, Float> particles, InteractionTypeArray<DIM, Float, Mode> interactions,
								   const Float softening, const Float theta,
								   size_t *bfsStackCounters, NodeArray<DIM, Float> bfsStackBuffers, const size_t stackCapacity) {
	
	/*
	if(blockIdx.x == 0 && threadIdx.x == 0){
		printf("Validating %lu groups @ (%p %p)\n", nGroups, groupInfo.childStart, groupInfo.childCount);
		for(size_t i = 0; i < nGroups; i++){
			printf("\t(%lu %lu)",groupInfo.childStart[i], groupInfo.childCount[i]);
		} printf("\n\n");
		printf("Validating tree\n");
		for(size_t i = 0; i < MAX_LEVELS; i++){
			printf("Layer %lu: has %lu @ (%p %p %p)\n", i, treeCounts[i], treeLevels[i].childStart, treeLevels[i].childCount, treeLevels[i].isLeaf);
			for(size_t j = 0; j < treeCounts[i]; j++){
				printf("\t(%lu %lu %d)",treeLevels[i].childStart[j], treeLevels[i].childCount[j], treeLevels[i].isLeaf[j]);
			} printf("\n\n");
		} printf("\n");
	}

	__threadfence();
	__syncthreads();

	//*/
	
	/*
	__threadfence();
	__syncthreads();
	if(blockIdx.x == 0 && threadIdx.x == 0){
	
		printf("Validating particles\n");
		printf("%p ",particles.m);
		for(size_t j = 0; j < DIM; j++){
			printf("%p ", particles.pos.x[j]);
		}
		for(size_t j = 0; j < DIM; j++){
			printf("%p ", particles.vel.x[j]);
		} printf("\n(");
		for(size_t i = 0; i < n; i++){
			printf("(%f ",particles.m[i]);
			for(size_t j = 0; j < DIM; j++){
				printf("%f ", particles.pos[i].x[j]);
			}
			for(size_t j = 0; j < DIM; j++){
				printf("%f ", particles.vel[i].x[j]);
			} printf(")\t");
		}printf("\n\n");
	
	}
	 //*/
	
	
#define BUF_MUL 128
	
	__shared__ size_t interactionCounters[1];
	__shared__ Float pointMass[BUF_MUL*INTERACTION_THRESHOLD];
	__shared__ Float pointPos[DIM * BUF_MUL*INTERACTION_THRESHOLD];
	
	
	if(threadIdx.x == 0 && blockIdx.x == 0){
		printf("%p %p %p\n",interactionCounters, pointMass, pointPos);
	}
	
	
	PointMassArray<DIM, Float> interactionList;
	interactionList.m = pointMass;
	for(size_t j = 0; j < DIM; j++){
		interactionList.pos.x[j] = pointPos + (j * BUF_MUL*INTERACTION_THRESHOLD);
	}
	interactionList.setCapacity(BUF_MUL*INTERACTION_THRESHOLD);
	
	if(threadIdx.x == 0 && blockIdx.x == 0){
		printf("%p\n",interactionList.m);
		for(size_t j = 0; j < DIM; j++){
			printf("%p ", interactionList.pos.x[j]);
		} printf("\n");
	}
	//if(threadIdx.x == 0)printf("%3d checking in (0)\n",blockIdx.x);
	
	for(size_t groupOffset = 0; groupOffset + blockIdx.x < nGroups; groupOffset += gridDim.x){
		if(blockIdx.x == 0 && threadIdx.x == 0) printf("%3d checking in with offset %lu / %lu, inc by %d\n",blockIdx.x,groupOffset,nGroups,gridDim.x);
		GroupInfo<DIM, Float, PPG> tgInfo;
		groupInfo.get(blockIdx.x + groupOffset,tgInfo);
		size_t threadsPerPart = blockDim.x / tgInfo.childCount;
		
		
		size_t* pGLCt = interactionCounters;
		PointMassArray<DIM, Float> pGList = interactionList;
		PointMassArray<DIM, Float> dummyP;
		initStack<PointMass,PointMassArray>(dummyP, 0, pGList, pGLCt);
		__threadfence_block();
		//if(threadIdx.x == 0) printf("pGLCt = %lu\n",*pGLCt);
		
		/*
		if(threadIdx.x == 0 && blockIdx.x == 0){
			printf("%p %p\n",pGLCt, pGList.m);
			for(size_t j = 0; j < DIM; j++){
				printf("%p ", pGList.pos.x[j]);
			} printf("\n");
		}
		//*/
		
		size_t* cLCt = bfsStackCounters + 2 * blockIdx.x;
		NodeArray<DIM, Float> currentLevel = bfsStackBuffers + 2 * blockIdx.x * stackCapacity;
		currentLevel.setCapacity(stackCapacity);
		__threadfence_block();
		/*
		if(threadIdx.x == 0 && blockIdx.x == 0){
			printf("%p %p %p %p\n",cLCt, currentLevel.isLeaf, currentLevel.childCount, currentLevel.childStart);
			for(size_t j = 0; j < DIM; j++){
				printf("%p ", currentLevel.minX.x[j]);
			} printf("\n");
			for(size_t j = 0; j < DIM; j++){
				printf("%p ", currentLevel.maxX.x[j]);
			} printf("\n");
			for(size_t j = 0; j < DIM; j++){
				printf("%p ", currentLevel.barycenter.pos.x[j]);
			} printf("\n");
			printf("%p %p\n",currentLevel.barycenter.m, currentLevel.radius);
		}
		//*/
		
		
		/*
		if(threadIdx.x == 0 && blockIdx.x == 0){
			printf("%d initing stack: %lu \n", blockIdx.x, startDepth);
			printf("%d continues: %p %p\n",blockIdx.x,treeLevels[startDepth].childCount, treeLevels[startDepth].childStart);
			printf("%d contains: %lu %lu\n",blockIdx.x,treeLevels[startDepth].childCount[0], treeLevels[startDepth].childStart[0]);
			//treeLevels[startDepth].childCount[0], treeLevels[startDepth].childStart[0]
		}
		//*/
		initStack<Node,NodeArray>(treeLevels[startDepth], treeCounts[startDepth], currentLevel, cLCt);
		
		
		size_t* nLCt = bfsStackCounters + 2 * blockIdx.x + 1;
		NodeArray<DIM, Float> nextLevel = bfsStackBuffers + (2 * blockIdx.x + 1) * stackCapacity;
		nextLevel.setCapacity(stackCapacity);
		/*
		if(threadIdx.x == 0 && blockIdx.x == 0){
			printf("%p %p %p %p\n",nLCt, nextLevel.isLeaf, nextLevel.childCount, nextLevel.childStart);
			for(size_t j = 0; j < DIM; j++){
				printf("%p ", nextLevel.minX.x[j]);
			} printf("\n");
			for(size_t j = 0; j < DIM; j++){
				printf("%p ", nextLevel.maxX.x[j]);
			} printf("\n");
			for(size_t j = 0; j < DIM; j++){
				printf("%p ", nextLevel.barycenter.pos.x[j]);
			} printf("\n");
			printf("%p %p\n",nextLevel.barycenter.m, nextLevel.radius);
		}
		//*/

		/*
		__syncthreads();
		if(threadIdx.x == 0 && blockIdx.x == 0){
			printf("%d post init stack: %lu \n", blockIdx.x, startDepth);
			printf("%d continues: %p %p\n",blockIdx.x,currentLevel.childCount, currentLevel.childStart);
			printf("%d contains: %lu %lu\n",blockIdx.x,currentLevel.childCount[0], currentLevel.childStart[0]);
			//treeLevels[startDepth].childCount[0], treeLevels[startDepth].childStart[0]
		}
		//*/
		__syncthreads();
		
		
		
		const size_t useful_thread_ct =  threadsPerPart * tgInfo.childCount;
		Particle<DIM, Float> particle;
		if(threadIdx.x < useful_thread_ct){
			if(tgInfo.childStart + (threadIdx.x % tgInfo.childCount) >= particles.elems){
				printf("Getting particle, %d < %lu, so want at %lu + (%d %% %lu) = %lu\n",threadIdx.x,useful_thread_ct,tgInfo.childStart, threadIdx.x, tgInfo.childCount, tgInfo.childStart + (threadIdx.x % tgInfo.childCount));
			}
			particles.get(tgInfo.childStart + (threadIdx.x % tgInfo.childCount), particle);
		}
		
		InteractionType<DIM, Float, Mode> interaction = freshInteraction<DIM, Float, Mode>();
		size_t curDepth = startDepth;
		while(*cLCt != 0 ){//&& curDepth < MAX_LEVELS){ // Second condition shouldn't matter....
			//if(threadIdx.x == 0) printf("%3d.%d Entering the land of disturbing loops\n",blockIdx.x,threadIdx.x);
			if(threadIdx.x == 0){
				*nLCt = 0;
			}
			/*
			if(threadIdx.x == 0 && blockIdx.x == 0){
				printf("%p %p %p %p\n",cLCt, currentLevel.isLeaf, currentLevel.childCount, currentLevel.childStart);
				for(size_t j = 0; j < DIM; j++){
					printf("%p ", currentLevel.minX.x[j]);
				} printf("\n");
				for(size_t j = 0; j < DIM; j++){
					printf("%p ", currentLevel.maxX.x[j]);
				} printf("\n");
				for(size_t j = 0; j < DIM; j++){
					printf("%p ", currentLevel.barycenter.pos.x[j]);
				} printf("\n");
				printf("%p %p\n",currentLevel.barycenter.m, currentLevel.radius);
				printf("%p %p %p %p\n",nLCt, nextLevel.isLeaf, nextLevel.childCount, nextLevel.childStart);
				for(size_t j = 0; j < DIM; j++){
					printf("%p ", nextLevel.minX.x[j]);
				} printf("\n");
				for(size_t j = 0; j < DIM; j++){
					printf("%p ", nextLevel.maxX.x[j]);
				} printf("\n");
				for(size_t j = 0; j < DIM; j++){
					printf("%p ", nextLevel.barycenter.pos.x[j]);
				} printf("\n");
				printf("%p %p\n",nextLevel.barycenter.m, nextLevel.radius);
			}
			//*/
			
			__threadfence_block();
			__syncthreads();
			
			ptrdiff_t startOfs = *cLCt;
			while(startOfs > 0){
				//if(threadIdx.x == 0) printf("\t%3d.%d Entering the inner crazy loop\n",blockIdx.x,threadIdx.x);
				ptrdiff_t toGrab = startOfs - blockDim.x + threadIdx.x;
				if(toGrab >= 0){
					Node<DIM, Float> nodeHere;
					currentLevel.get(toGrab, nodeHere);
					//if(threadIdx.x == 0) printf("\t%d.%d @ %lu:\t%lu %lu vs %lu %lu with %lu %ld \n", blockIdx.x, threadIdx.x, curDepth, nodeHere.childStart, nodeHere.childCount, currentLevel.childStart[toGrab], currentLevel.childCount[toGrab], *cLCt, toGrab);
					//*
					if(passesMAC(tgInfo, nodeHere, theta)){
						//if(threadIdx.x == 0) printf("\t%d accepted MAC\n",threadIdx.x);
						if(INTERACTION_THRESHOLD > 0){
							// Store to C/G list
							//if(threadIdx.x == 0) printf("\t%d found the following:\n\t(%lu %lu) @ (%p/%lu), writing to stack at pos %lu @ %p\n", threadIdx.x, nodeHere.childStart, nodeHere.childCount, treeLevels[curDepth + 1].childStart, curDepth + 1, *nLCt,nLCt);
							
							PointMassArray<DIM, Float> tmpArray(nodeHere.barycenter);
							size_t tmpCt = 1;
							
							pushAll<PointMass,PointMassArray>(tmpArray, tmpCt, pGList, pGLCt);
						} else if(threadIdx.x < tgInfo.childCount){
							//interaction = interaction + calc_force(particle.m, particle.pos, nodeHere.mass, nodeHere.barycenter, softening);
						}
					} else {
						//if(threadIdx.x == 0) printf("\t%d rejected MAC\n",threadIdx.x);
						if(nodeHere.isLeaf){
							if(INTERACTION_THRESHOLD > 0){
								// Store to P/G list
								//printf("Pushing particles %lu particles, ")
								if(nodeHere.childCount > 16){
									printf("\t%d.%d: Adding a lot particles %lu\n",blockIdx.x,threadIdx.x,nodeHere.childCount);
								}
								pushAll<PointMass,PointMassArray>(particles.mass + nodeHere.childStart, nodeHere.childCount, pGList, pGLCt);
							} else {
								/*
								 for(size_t pI = nodeHere.childCount; pI > 0; pI -= threadsPerPart ){
									ptrdiff_t toGrab = pI - threadsPerPart + (threadIdx.x / tgInfo.childCount);
									if(toGrab >= 0){
								 interaction = interaction + calc_force(particle.m, particle.pos, particles[nodeHere.childStart + toGrab].m, particles[nodeHere.childStart + toGrab].pos, softening);
									}
								 }
								 */
							}
						} else {
							pushAll<Node, NodeArray>(treeLevels[curDepth + 1] + nodeHere.childStart, nodeHere.childCount, nextLevel, nLCt);
						}
					}
					//*/
				}
				__threadfence_block();
				__syncthreads();
				if(threadIdx.x == 0)  {
				//	printf("\t%3d.%d All safely past toGrab\n",blockIdx.x,threadIdx.x);
				}
				
				/*
				if(threadIdx.x == 0 && blockIdx.x == 0){
					printf("%p %p %p %p\n",cLCt, currentLevel.isLeaf, currentLevel.childCount, currentLevel.childStart);
					for(size_t j = 0; j < DIM; j++){
						printf("%p ", currentLevel.minX.x[j]);
					} printf("\n");
					for(size_t j = 0; j < DIM; j++){
						printf("%p ", currentLevel.maxX.x[j]);
					} printf("\n");
					for(size_t j = 0; j < DIM; j++){
						printf("%p ", currentLevel.barycenter.pos.x[j]);
					} printf("\n");
					printf("%p %p\n",currentLevel.barycenter.m, currentLevel.radius);
					printf("%p %p %p %p\n",nLCt, nextLevel.isLeaf, nextLevel.childCount, nextLevel.childStart);
					for(size_t j = 0; j < DIM; j++){
						printf("%p ", nextLevel.minX.x[j]);
					} printf("\n");
					for(size_t j = 0; j < DIM; j++){
						printf("%p ", nextLevel.maxX.x[j]);
					} printf("\n");
					for(size_t j = 0; j < DIM; j++){
						printf("%p ", nextLevel.barycenter.pos.x[j]);
					} printf("\n");
					printf("%p %p\n",nextLevel.barycenter.m, nextLevel.radius);
				}
				//*/
				
				
				//*
				if(INTERACTION_THRESHOLD > 0){ // Can't diverge, compile-time constant
					ptrdiff_t innerStartOfs;
					//if(threadIdx.x == 0) printf("\t%d PGLCt is %lu >? %lu (%ld > %ld)\n",threadIdx.x,*pGLCt,INTERACTION_THRESHOLD,(ptrdiff_t)(*pGLCt),(ptrdiff_t)INTERACTION_THRESHOLD);
					for(innerStartOfs = *pGLCt; innerStartOfs >= (ptrdiff_t)INTERACTION_THRESHOLD; innerStartOfs -= threadsPerPart){
						ptrdiff_t toGrab = innerStartOfs - threadsPerPart + (threadIdx.x / tgInfo.childCount);
						// printf("\t%d interacting with %ld = %lu - %lu + (%d / %d)\n",threadIdx.x,toGrab,innerStartOfs,threadsPerPart,threadIdx.x,tgInfo.childCount);
						if(toGrab >= 0){
							PointMass<DIM, Float> pHere;
							pGList.get(toGrab, pHere);
							interaction = interaction + calc_force(particle.mass, pHere, softening);
						}
					}
					//if(threadIdx.x == 0) printf("\t%d through interaction loop safely\n",threadIdx.x);
					// Need to update stack pointer
					// Need to update stack pointer
					if(threadIdx.x == 0){
						atomicExch((unsigned long long *)pGLCt, (unsigned long long)((innerStartOfs < 0) ? 0 : innerStartOfs));
					}
				 
				}
				//*/
				
				//if(threadIdx.x == 0) printf("%3d.%d: Try going around again\n",blockIdx.x,threadIdx.x);
				
				startOfs -= blockDim.x;
				
				
			}
			
			//if(threadIdx.x == 0) printf("%3d.%d Done inside: %lu (loopcount at %ld) work remaining at depth: %lu\n",blockIdx.x, threadIdx.x, *nLCt,startOfs,curDepth);
			
			swap<NodeArray<DIM, Float>>(currentLevel, nextLevel);
			swap<size_t*>(cLCt, nLCt);
			curDepth += 1;
		}
		
		// Process remaining interactions
		//printf("Time to process remainder\n");

		//*

		__threadfence_block();
		__syncthreads();

		if(INTERACTION_THRESHOLD > 0){ // Can't diverge, compile-time constant
			ptrdiff_t innerStartOfs;

			for(innerStartOfs = *pGLCt; innerStartOfs > 0; innerStartOfs -= threadsPerPart){
				ptrdiff_t toGrab = innerStartOfs - threadsPerPart + (threadIdx.x / tgInfo.childCount);
				if(toGrab >= 0){
					PointMass<DIM, Float> pHere;
					pGList.get(toGrab, pHere);
					interaction = interaction + calc_force(particle.mass, pHere, softening);
				}
			}
			// Need to update stack pointer
			// Need to update stack pointer
			if(threadIdx.x == 0){
				atomicExch((unsigned long long *)pGLCt, 0);
			}

		}

		// This needs to be done in shared memory! We should figure out how to combine with the stack scratch-space!

		if(threadIdx.x < useful_thread_ct){
			interactions.set(tgInfo.childStart + threadIdx.x, interaction);
		}
		//printf("Remainder processed\n");

		__threadfence_block();
		__syncthreads(); // All forces have been summed and are in view

		// reduce (hack-job fashion for now) if multithreading per particle in play
		//*
		//printf("Reducing\n");
		if(threadIdx.x < tgInfo.childCount){
			InteractionType<DIM, Float, Mode> accInt = freshInteraction<DIM, Float, Mode>();
			for(size_t i = 1; i < threadsPerPart; i++){
				InteractionType<DIM, Float, Mode> tmp;
				interactions.get(tgInfo.childStart + threadIdx.x + i * tgInfo.childCount, tmp);
				accInt = accInt + tmp;
			}
			interactions.set(tgInfo.childStart + threadIdx.x, interaction + accInt);

		}
		//if(threadIdx.x == 0) printf("%3d Done reducing\n",blockIdx.x);
		//*/
		
	}
	return;
	
}


// Something is badly wrong with template resolution if we switch to InteractionType here.
// I think the compilers are doing name-mangling differently or something
template<size_t DIM, typename Float, size_t PPG, size_t MAX_LEVELS, size_t MAX_STACK_ENTRIES, size_t INTERACTION_THRESHOLD, TraverseMode Mode>
void traverseTreeCUDA(size_t nGroups, GroupInfoArray<DIM, Float, PPG> groupInfo, size_t startDepth,
					  NodeArray<DIM, Float> treeLevels[MAX_LEVELS], size_t treeCounts[MAX_LEVELS], size_t n, ParticleArray<DIM, Float> particles, VecArray<DIM, Float> interactions, Float softening, Float theta, size_t blockCt, size_t threadCt){

	std::cout << "Traverse tree with " << blockCt << " blocks and " << threadCt << " tpb"<<std::endl;
	NodeArray<DIM, Float> placeHolderLevels[MAX_LEVELS];
	makeDeviceTree<DIM, Float, MAX_LEVELS>(treeLevels, placeHolderLevels, treeCounts);
	NodeArray<DIM, Float>* cuTreeLevels;

	ALLOC_DEBUG_MSG(MAX_LEVELS*sizeof(NodeArray<DIM, Float>) + MAX_LEVELS * sizeof(size_t));

	gpuErrchk( (hipMalloc(&cuTreeLevels, MAX_LEVELS*sizeof(NodeArray<DIM, Float>))) );
	gpuErrchk( (hipMemcpy(cuTreeLevels, placeHolderLevels, MAX_LEVELS*sizeof(NodeArray<DIM, Float>), hipMemcpyHostToDevice)) );
	
	size_t* cuTreeCounts;
	gpuErrchk( (hipMalloc(&cuTreeCounts, MAX_LEVELS * sizeof(size_t))) );
	gpuErrchk( (hipMemcpy(cuTreeCounts, treeCounts, MAX_LEVELS * sizeof(size_t), hipMemcpyHostToDevice)) );
	
	
	size_t biggestRow = 0;
	for(size_t level = 0; level < MAX_LEVELS; level++){
		biggestRow = (treeCounts[level] > biggestRow) ? treeCounts[level] : biggestRow;
	}


	std::cout << "Biggest row: " << biggestRow  << std::endl;


	const size_t stackCapacity = biggestRow;
	const size_t blocksPerLaunch = MAX_STACK_ENTRIES / stackCapacity;
	std::cout << "Allowing: " << blocksPerLaunch << " blocks per launch" << std::endl;

	NodeArray<DIM, Float> bfsStackBuffers;
	size_t * bfsStackCounters;
	allocDeviceNodeArray(blocksPerLaunch * 2 * stackCapacity, bfsStackBuffers);

	ALLOC_DEBUG_MSG(blocksPerLaunch * 2 * sizeof(size_t));
	gpuErrchk( (hipMalloc(&bfsStackCounters, blocksPerLaunch * 2 * sizeof(size_t))) );
	
	
	GroupInfoArray<DIM, Float, PPG> cuGroupInfo;
	allocDeviceGroupInfoArray(nGroups, cuGroupInfo);
	copyDeviceGroupInfoArray(nGroups, cuGroupInfo, groupInfo, hipMemcpyHostToDevice);
	
	ParticleArray<DIM, Float> cuParticles;
	allocDeviceParticleArray(n, cuParticles);
	copyDeviceParticleArray(n, cuParticles, particles, hipMemcpyHostToDevice);
	
	InteractionTypeArray<DIM, Float, Mode> cuInteractions;
	allocDeviceVecArray(n, cuInteractions);
	copyDeviceVecArray(n, cuInteractions, interactions, hipMemcpyHostToDevice);
	
	dim3 dimGrid(blocksPerLaunch);
	dim3 dimBlock(threadCt);
	std::cout << "Trying to launch with " << threadCt << " / block with " << blocksPerLaunch << " blocks" << std::endl;
	
	traverseTreeKernel<DIM, Float, PPG, MAX_LEVELS, INTERACTION_THRESHOLD, Mode><<<dimGrid, dimBlock>>>(nGroups, cuGroupInfo, startDepth, cuTreeLevels, cuTreeCounts, n, cuParticles, cuInteractions, softening, theta, bfsStackCounters, bfsStackBuffers, stackCapacity);
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );
	
	copyDeviceVecArray(n, interactions, cuInteractions, hipMemcpyDeviceToHost);
	
	freeDeviceVecArray(cuInteractions);
	freeDeviceParticleArray(cuParticles);
	freeDeviceGroupInfoArray(cuGroupInfo);
	freeDeviceNodeArray(bfsStackBuffers);
	gpuErrchk( (hipFree(bfsStackCounters)) );
	gpuErrchk( (hipFree(cuTreeCounts)) );
	freeDeviceTree<DIM, Float, MAX_LEVELS>(placeHolderLevels);
	gpuErrchk( (hipFree(cuTreeLevels)) );
	
	
	
}

template void traverseTreeCUDA<3, float, 16, 16, 300000, 8, Forces>(size_t, GroupInfoArray<3, float, 16>, size_t, NodeArray<3, float> *, size_t *, size_t, ParticleArray<3, float>, InteractionTypeArray<3, float, Forces>, float, float, size_t, size_t);


