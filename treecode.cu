#include "hip/hip_runtime.h"
#include "treedefs.h"
#include "CudaArrayCopyUtils.h"
#include "treecodeCU.h"
#include "cudahelper.h"
#include "tictoc.h"
#include <iostream>

//: We should really be using native CUDA vectors for this.... but that requires more funny typing magic to convert the CPU data

template<our_size_t DIM, typename Float, our_size_t PPG> __device__ bool passesMAC(const GroupInfo<DIM, Float, PPG>& groupInfo, const Node<DIM, Float>& nodeHere, Float theta) {
	
	Float d = mag(groupInfo.center - nodeHere.barycenter.pos) - groupInfo.radius;
	Float l = 2 * nodeHere.radius;
	return d > (l / theta);
	
}


template<template<our_size_t, typename> class ElemType, template<our_size_t, typename> class ElemTypeArray, our_size_t DIM, typename Float>
__device__ void initStack(ElemTypeArray<DIM, Float> level, our_size_t levelCt, ElemTypeArray<DIM, Float> stack, our_size_t* stackCt){
	if(threadIdx.x < levelCt){
		if(levelCt > level.elems || threadIdx.x >= stack.elems) printf("%d.%d %s: " SZSTR " (should be <= " SZSTR ") elements. stackCt @ %d / " SZSTR "\n",blockIdx.x, threadIdx.x, __func__,levelCt,level.elems,threadIdx.x,stack.elems);

		ElemType<DIM, Float> eHere;
		level.get(threadIdx.x, eHere);
		stack.set(threadIdx.x, eHere);
	}
	if(threadIdx.x == 0){
		atomicExch((cu_size_t*)stackCt,(cu_size_t)levelCt); // Don't want to have to threadfence afterwards. Just make sure it's set!
	}
}

template<our_size_t DIM, typename Float> __device__ void dumpStackChildren(NodeArray<DIM, Float> stack, const our_size_t* stackCt){
	our_size_t dst = *stackCt;
	for(our_size_t i = 0; i < dst; i++){
			printf("(" SZSTR ") see (" SZSTR " " SZSTR ") in (%p/" SZSTR ")\n",i,stack.childStart[i],stack.childCount[i], stackCt, dst);
	}
}

template<our_size_t DIM, typename T> __device__ void pushMeta(const our_size_t srcSt, const our_size_t srcCt, PointMassArray<DIM, T> stack, our_size_t* stackCt){
	// This is a weird compiler bug. There's no reason this shouldn't have worked without the cast.
	our_size_t dst = atomicAdd((cu_size_t*)stackCt, (cu_size_t)srcCt);
	for(our_size_t i = dst, j = 0; i < dst + srcCt; i++, j++){
		PointMass<DIM, T> eHere;
		eHere.m = 1;
		eHere.pos.x[0] = srcSt + j;
		eHere.pos.x[1] = 0;
		eHere.pos.x[2] = 0;
		stack.set(i, eHere);
	}
}

template<template<our_size_t, typename> class ElemType, template<our_size_t, typename> class ElemTypeArray, our_size_t DIM, typename Float> __device__ void pushAll(const ElemTypeArray<DIM, Float> src, const our_size_t srcCt, ElemTypeArray<DIM, Float> stack, our_size_t* stackCt){
	// This is a weird compiler bug. There's no reason this shouldn't have worked without the cast.
	our_size_t dst = atomicAdd((cu_size_t*)stackCt, (cu_size_t)srcCt);
	if(srcCt > src.elems || dst >= stack.elems)  printf("%d.%d %s: " SZSTR " (should be <= " SZSTR ") elements. stackCt @ " SZSTR " / " SZSTR "\n",blockIdx.x, threadIdx.x, __func__, srcCt,src.elems,dst,stack.elems);
	for(our_size_t i = dst, j = 0; i < dst + srcCt; i++, j++){
		ElemType<DIM, Float> eHere;
		src.get(j, eHere);
		stack.set(i, eHere);
	}
}

template<our_size_t DIM, typename Float, TraverseMode Mode> __device__ InteractionType(DIM, Float, Mode) freshInteraction(){
	InteractionType(DIM, Float, Mode) fresh; for(our_size_t i = 0; i < DIM; i++){
		fresh.x[i] = 0.0;
	}
	return fresh;
}

// Needs softening

template<our_size_t DIM, typename Float, TraverseMode Mode, bool spam = false> __device__ InteractionType(DIM, Float, Mode) calc_interaction(const PointMass<DIM, Float> &m1, const InteracterType(DIM, Float, Mode) &m2, Float softening){
	InteractionType(DIM, Float, Mode) interaction = freshInteraction<DIM, Float, Mode>();
	our_size_t isParticle = m2.m != 0;
	switch(Mode){
	case Forces:{
		// Reinterpret cast is evil, but necessary here. template-induced dead-code and all.
		if(spam){
			printf("Interacting\t%f %f %f %f vs %f %f %f %f\n",m1.m,m1.pos.x[0],m1.pos.x[1],m1.pos.x[2],m2.m,m2.pos.x[0],m2.pos.x[1],m2.pos.x[2]);
		}
		const PointMass<DIM, Float>& m2_inner = reinterpret_cast<const PointMass<DIM, Float>& >(m2);
		Vec<DIM, Float> disp = m1.pos - m2_inner.pos;
		interaction = (disp * ((m1.m * m2_inner.m) / (Float)(softening + pow((Float)mag_sq(disp),(Float)1.5))));
		break;}
	case CountOnly:
		interaction.x[isParticle] = 1;
		break;
	case HashInteractions:{
		// Type inference is failing here unless these are all explicitly separate variables
		our_size_t generic = m2.pos.x[0];
		our_size_t nodeSpecific1 = m2.pos.x[1];
		our_size_t nodeSpecific2 = m2.pos.x[2];
		our_size_t nodeSpecific = nodeSpecific1 ^ nodeSpecific2;
		our_size_t e = generic ^ (!isParticle) * nodeSpecific;
		interaction.x[isParticle] = e;
		break;}
	}
	return interaction;
}

template<typename T> __device__ inline void swap(T &a, T &b){
	T c(a);	a=b;	b=c;
}

template<our_size_t DIM, typename T> __device__ inline void dumpNodeArrayContents(const char *name, const NodeArray<DIM, T> &n){
	ASSERT_DEAD_CODE;
	printf("%s : %p %p %p %p " SZSTR "\n",name, n.isLeaf, n.childCount, n.childStart, n.radius, n.elems);
	for(our_size_t i = 0; i < DIM; i++){
		printf("%p ",n.minX.x[i]);
	}printf("\n");
	for(our_size_t i = 0; i < DIM; i++){
		printf("%p ",n.maxX.x[i]);
	}printf("\n");
	for(our_size_t i = 0; i < DIM; i++){
		printf("%p ",n.barycenter.pos.x[i]);
	}printf("%p\n",n.barycenter.m);
	printf("%s - Dump complete\n",name);
}


template<our_size_t DIM, typename Float, our_size_t TPB, our_size_t PPG, our_size_t MAX_LEVELS, our_size_t INTERACTION_THRESHOLD, TraverseMode Mode, bool spam>
__global__ void traverseTreeKernel(const our_size_t nGroups, const GroupInfoArray<DIM, Float, PPG> groupInfo,
								   const our_size_t startDepth, NodeArray<DIM, Float>* treeLevels, const our_size_t* treeCounts,
								   const our_size_t n, const ParticleArray<DIM, Float> particles, InteractionTypeArray(DIM, Float, Mode) interactions,
								   const Float softening, const Float theta,
								   our_size_t *bfsStackCounters, NodeArray<DIM, Float> bfsStackBuffers, const our_size_t stackCapacity) {
typedef typename std::conditional<NonForceCondition(Mode), our_size_t, Float>::type InteractionElemType;
#define BUF_MUL 128
	// TPB must = blockDims.x
	__shared__ our_size_t interactionCounters[1];
	__shared__ InteractionElemType pointMass[BUF_MUL*INTERACTION_THRESHOLD];
	__shared__ InteractionElemType pointPos[DIM * BUF_MUL*INTERACTION_THRESHOLD];
	__shared__ InteractionElemType interactionBuf[TPB * InteractionElems(Mode, DIM, 2)];
	
	if(threadIdx.x == 0 && blockDim.x != TPB){
		printf("%d Launched with mismatched TPB parameters\n",blockIdx.x);
	}

	InteracterTypeArray(DIM, Float, Mode) interactionList;
	interactionList.m = pointMass;
	for(our_size_t j = 0; j < DIM; j++){
		interactionList.pos.x[j] = pointPos + (j * BUF_MUL*INTERACTION_THRESHOLD);
	}
	interactionList.setCapacity(BUF_MUL*INTERACTION_THRESHOLD);
	
	InteractionTypeArray(DIM, Float, Mode) interactionScratch;
	for(our_size_t j = 0; j < InteractionElems(Mode, DIM, 2); j++){
		interactionScratch.x[j] = interactionBuf + (TPB * j);
	}
	interactionScratch.setCapacity(TPB);

	
	for(our_size_t groupOffset = 0; groupOffset + blockIdx.x < nGroups; groupOffset += gridDim.x){
		GroupInfo<DIM, Float, PPG> tgInfo;
		groupInfo.get(blockIdx.x + groupOffset,tgInfo);
		our_size_t threadsPerPart = blockDim.x / tgInfo.childCount;
		
		
		our_size_t* pGLCt = interactionCounters;
		InteracterTypeArray(DIM, Float, Mode) pGList = interactionList;
		InteracterTypeArray(DIM, Float, Mode) dummyP;
		initStack<PointMass,PointMassArray>(dummyP, 0, pGList, pGLCt);
		
		our_size_t* cLCt = bfsStackCounters + 2 * blockIdx.x;
		NodeArray<DIM, Float> currentLevel = bfsStackBuffers + 2 * blockIdx.x * stackCapacity;
		currentLevel.setCapacity(stackCapacity);
		initStack<Node,NodeArray>(treeLevels[startDepth], treeCounts[startDepth], currentLevel, cLCt);
		
		
		our_size_t* nLCt = bfsStackCounters + 2 * blockIdx.x + 1;
		NodeArray<DIM, Float> nextLevel = bfsStackBuffers + (2 * blockIdx.x + 1) * stackCapacity;
		nextLevel.setCapacity(stackCapacity);

		__threadfence_block();
		__syncthreads(); // Everyone needs the stack initialized before we can continue
		
		
		
		const our_size_t useful_thread_ct =  threadsPerPart * tgInfo.childCount;
		Particle<DIM, Float> particle;
		if(threadIdx.x < useful_thread_ct){
			if(tgInfo.childStart + (threadIdx.x % tgInfo.childCount) >= particles.elems){
				printf("Getting particle, %d < " SZSTR ", so want at " SZSTR " + (%d %% " SZSTR ") = " SZSTR "\n",threadIdx.x,useful_thread_ct,tgInfo.childStart, threadIdx.x, tgInfo.childCount, tgInfo.childStart + (threadIdx.x % tgInfo.childCount));
			}
			particles.get(tgInfo.childStart + (threadIdx.x % tgInfo.childCount), particle);
		}
		
		InteractionType(DIM, Float, Mode) interaction = freshInteraction<DIM, Float, Mode>();
		our_size_t curDepth = startDepth;
		while(*cLCt != 0 ){
			if(threadIdx.x == 0){
				*nLCt = 0;
				printf("%d+" SZSTR " Traversing level " SZSTR "\n",blockIdx.x,groupOffset,curDepth);
			}
			
			__threadfence_block();
			__syncthreads();
			
			cu_diff_t startOfs = *cLCt;
			if(spam && threadIdx.x == 0){
				printf("" SZSTR "." SZSTR " has " DFSTR " @ " SZSTR "\n",blockIdx.x + groupOffset,tgInfo.childStart + (threadIdx.x % tgInfo.childCount), startOfs,	curDepth);
			}

			while(startOfs > 0){
				cu_diff_t toGrab = startOfs - blockDim.x + threadIdx.x;
				if(toGrab >= 0){
					Node<DIM, Float> nodeHere;
					currentLevel.get(toGrab, nodeHere);
					//if(threadIdx.x == 0) printf("\t%d.%d @ " SZSTR ":\t" SZSTR " " SZSTR " vs " SZSTR " " SZSTR " with " SZSTR " " DFSTR " \n", blockIdx.x, threadIdx.x, curDepth, nodeHere.childStart, nodeHere.childCount, currentLevel.childStart[toGrab], currentLevel.childCount[toGrab], *cLCt, toGrab);
					//*
					if(spam){
						printf("" SZSTR "." SZSTR " comparing against node @ " SZSTR "." SZSTR ":" SZSTR ".%d = %d\n",
								blockIdx.x + groupOffset,
								tgInfo.childStart + (threadIdx.x % tgInfo.childCount),
								curDepth,nodeHere.childStart,nodeHere.childCount,nodeHere.isLeaf,passesMAC<DIM, Float, PPG>(tgInfo, nodeHere, theta));
					}
					if(passesMAC(tgInfo, nodeHere, theta)){
						//if(threadIdx.x == 0) printf("\t%d accepted MAC\n",threadIdx.x);
						if(INTERACTION_THRESHOLD > 0){
							// Store to C/G list
							//if(threadIdx.x == 0) printf("\t%d found the following:\n\t(" SZSTR " " SZSTR ") @ (%p/" SZSTR "), writing to stack at pos " SZSTR " @ %p\n", threadIdx.x, nodeHere.childStart, nodeHere.childCount, treeLevels[curDepth + 1].childStart, curDepth + 1, *nLCt,nLCt);
							
							InteracterType(DIM, Float, Mode) nodePush;
							switch(Mode){
							case Forces:{
									nodePush = nodeHere.barycenter; break;}
							case CountOnly:
							case HashInteractions:{
									nodePush.m = 0;
									nodePush.pos.x[0] = curDepth;
									nodePush.pos.x[1] = nodeHere.childStart;
									nodePush.pos.x[2] = nodeHere.childCount;
									break;}
							}
							InteracterTypeArray(DIM, Float, Mode) tmpArray(nodePush);
							our_size_t tmpCt = 1;
							
							pushAll<PointMass,PointMassArray>(tmpArray, tmpCt, pGList, pGLCt);
						} else if(threadIdx.x < tgInfo.childCount){
							ASSERT_DEAD_CODE;
							//interaction = interaction + calc_force(particle.m, particle.pos, nodeHere.mass, nodeHere.barycenter, softening);
						}
					} else {
						//if(threadIdx.x == 0) printf("\t%d rejected MAC\n",threadIdx.x);
						if(nodeHere.isLeaf){
							if(INTERACTION_THRESHOLD > 0){
								// Store to P/G list
								//printf("Pushing particles " SZSTR " particles, ")
								if(spam) printf("" SZSTR "." SZSTR " leaf contains " SZSTR ":" SZSTR "\n",blockIdx.x + groupOffset, tgInfo.childStart + (threadIdx.x % tgInfo.childCount),nodeHere.childStart,nodeHere.childCount);

								if(nodeHere.childCount > 16){
									printf("\t%d.%d: Adding a lot particles " SZSTR "\n",blockIdx.x,threadIdx.x,nodeHere.childCount);
								}
								switch(Mode){
								case Forces:{
									pushAll<PointMass, PointMassArray>(particles.mass + nodeHere.childStart, nodeHere.childCount, *reinterpret_cast<PointMassArray<DIM, Float>* >(&pGList), pGLCt); break;}
								case CountOnly:
								case HashInteractions:{
									pushMeta(nodeHere.childStart, nodeHere.childCount, pGList, pGLCt);
									break;}
								}

							} else {
								ASSERT_DEAD_CODE;
								/*
								 for(our_size_t pI = nodeHere.childCount; pI > 0; pI -= threadsPerPart ){
									cu_diff_t toGrab = pI - threadsPerPart + (threadIdx.x / tgInfo.childCount);
									if(toGrab >= 0){
								 interaction = interaction + calc_force(particle.m, particle.pos, particles[nodeHere.childStart + toGrab].m, particles[nodeHere.childStart + toGrab].pos, softening);
									}
								 }
								 */
							}
						} else {
							pushAll<Node, NodeArray>(treeLevels[curDepth + 1] + nodeHere.childStart, nodeHere.childCount, nextLevel, nLCt);
						}
					}
					//*/
				}
				__threadfence_block();
				__syncthreads();
				// if(threadIdx.x == 0) printf("\t%3d.%d All safely past toGrab\n",blockIdx.x,threadIdx.x);
				
				//*
				if(INTERACTION_THRESHOLD > 0){ // Can't diverge, compile-time constant
					cu_diff_t innerStartOfs = 0;
					/*
					if(threadIdx.x == 0) printf("\t%d PGLCt is " SZSTR " >? " SZSTR " (" DFSTR " > " DFSTR ")\n",threadIdx.x,*pGLCt,INTERACTION_THRESHOLD,(cu_diff_t)(*pGLCt),(cu_diff_t)INTERACTION_THRESHOLD);

					// The casting here feels very strange - why is innerStartOfs implicitly casted, rather than vice versa?
					for(innerStartOfs = *pGLCt; innerStartOfs >= (cu_diff_t)INTERACTION_THRESHOLD; innerStartOfs -= threadsPerPart){
						cu_diff_t toGrab = innerStartOfs - threadsPerPart + (threadIdx.x / tgInfo.childCount);
						if(toGrab >= 0 && threadIdx.x < useful_thread_ct){
							if(toGrab % threadsPerPart == 0) printf("\t%d interacting with " DFSTR " = " SZSTR " - " SZSTR " + (%d / %d)\n",threadIdx.x,toGrab,innerStartOfs,threadsPerPart,threadIdx.x,tgInfo.childCount);
							InteracterType(DIM, Float, Mode) pHere;
							pGList.get(toGrab, pHere);
							interaction = interaction + calc_interaction<DIM, Float, Mode, spam>(particle.mass, pHere, softening);
						}
					}
					if(threadIdx.x == 0) printf("\t%d through interaction loop safely\n",threadIdx.x);
					// Need to update stack pointer
					//*/
					if(threadIdx.x == 0){
						atomicExch((cu_size_t *)pGLCt, (cu_size_t)((innerStartOfs < 0) ? 0 : innerStartOfs));
					}
				}
				//*/

				//if(threadIdx.x == 0) printf("%3d.%d: Try going around again\n",blockIdx.x,threadIdx.x);
				startOfs -= blockDim.x;
			}
			
			//if(threadIdx.x == 0) printf("%3d.%d Done inside: " SZSTR " (loopcount at " DFSTR ") work remaining at depth: " SZSTR "\n",blockIdx.x, threadIdx.x, *nLCt,startOfs,curDepth);
			swap<NodeArray<DIM, Float>>(currentLevel, nextLevel);
			swap<our_size_t*>(cLCt, nLCt);
			curDepth += 1;
		}
		
		// Process remaining interactions
		//*
		__threadfence_block();
		__syncthreads();

		if(INTERACTION_THRESHOLD > 0){ // Can't diverge, compile-time constant
			cu_diff_t innerStartOfs = 0;
			/*
			if(threadIdx.x == 0) printf("\t%d PGLCt is " SZSTR " >? " SZSTR " (" DFSTR " > " DFSTR ")\n",threadIdx.x,*pGLCt,INTERACTION_THRESHOLD,(cu_diff_t)(*pGLCt),(cu_diff_t)INTERACTION_THRESHOLD);

			for(innerStartOfs = *pGLCt; innerStartOfs > 0; innerStartOfs -= threadsPerPart){
				cu_diff_t toGrab = innerStartOfs - threadsPerPart + (threadIdx.x / tgInfo.childCount);
				if(toGrab >= 0 && threadIdx.x < useful_thread_ct){
					if(toGrab % threadsPerPart == 0) printf("\t%d interacting with " DFSTR " = " SZSTR " - " SZSTR " + (%d / %d)\n",threadIdx.x,toGrab,innerStartOfs,threadsPerPart,threadIdx.x,tgInfo.childCount);
					InteracterType(DIM, Float, Mode) pHere;
					pGList.get(toGrab, pHere);
					interaction = interaction + calc_interaction<DIM, Float, Mode, spam>(particle.mass, pHere, softening);
				}
			}
			if(threadIdx.x == 0) printf("\t%d through final interaction loop safely\n",threadIdx.x);
			//*/
			if(threadIdx.x == 0){
				atomicExch((cu_size_t *)pGLCt, 0);
			}
		}

		// This needs to be done in shared memory! We should figure out how to combine with the stack scratch-space!
		if(threadIdx.x < useful_thread_ct){
			interactionScratch.set(threadIdx.x, interaction);
		}

		__threadfence_block();
		__syncthreads(); // All forces have been summed and are in view

		// reduce (hack-job fashion for now) if multithreading per particle in play
		//*
		//printf("Reducing\n");
		if(threadIdx.x < tgInfo.childCount){
			InteractionType(DIM, Float, Mode) accInt = freshInteraction<DIM, Float, Mode>();
			for(our_size_t i = 1; i < threadsPerPart; i++){
				InteractionType(DIM, Float, Mode) tmp;
				interactionScratch.get(threadIdx.x + i * tgInfo.childCount, tmp);
				accInt = accInt + tmp;
			}
			interactions.set(tgInfo.childStart + threadIdx.x, interaction + accInt);

		}
		//if(threadIdx.x == 0) printf("%3d Done reducing\n",blockIdx.x);
		//*/
		
	}
	return;
	
}


// Something is badly wrong with template resolution if we switch to InteractionType here.
// I think the compilers are doing name-mangling differently or something


/*
 template void traverseTreeCUDA<3, float, 128, 16, 16, 300000, 8, Forces>
 	 	 	 	 	 (our_size_t, GroupInfoArray<3, float, 16>, our_size_t,
 	 	 	 	 	 NodeArray<3, float> *, our_size_t *,
 	 	 	 	 	 our_size_t, ParticleArray<3, float>, VecArray<3, float>, float, float, our_size_t);
*/
template<our_size_t DIM, typename Float, our_size_t threadCt, our_size_t PPG, our_size_t MAX_LEVELS, our_size_t MAX_STACK_ENTRIES, our_size_t INTERACTION_THRESHOLD, TraverseMode Mode, bool spam>
void traverseTreeCUDA(our_size_t nGroups, GroupInfoArray<DIM, Float, PPG> groupInfo, our_size_t startDepth,
					  NodeArray<DIM, Float> treeLevels[MAX_LEVELS], our_size_t treeCounts[MAX_LEVELS],
					  our_size_t n, ParticleArray<DIM, Float> particles, InteractionTypeArray(DIM, Float, Mode) interactions, Float softening, Float theta, our_size_t blockCt){

	std::cout << "Traverse tree with " << blockCt << " blocks and " << threadCt << " tpb"<<std::endl;
	NodeArray<DIM, Float> placeHolderLevels[MAX_LEVELS];
	makeDeviceTree<DIM, Float, MAX_LEVELS>(treeLevels, placeHolderLevels, treeCounts);
	NodeArray<DIM, Float>* cuTreeLevels;

	ALLOC_DEBUG_MSG(MAX_LEVELS*sizeof(NodeArray<DIM, Float>) + MAX_LEVELS * sizeof(our_size_t));

	gpuErrchk( (hipMalloc(&cuTreeLevels, MAX_LEVELS*sizeof(NodeArray<DIM, Float>))) );
	gpuErrchk( (hipMemcpy(cuTreeLevels, placeHolderLevels, MAX_LEVELS*sizeof(NodeArray<DIM, Float>), hipMemcpyHostToDevice)) );
	
	our_size_t* cuTreeCounts;
	gpuErrchk( (hipMalloc(&cuTreeCounts, MAX_LEVELS * sizeof(our_size_t))) );
	gpuErrchk( (hipMemcpy(cuTreeCounts, treeCounts, MAX_LEVELS * sizeof(our_size_t), hipMemcpyHostToDevice)) );
	
	
	our_size_t biggestRow = 0;
	for(our_size_t level = 0; level < MAX_LEVELS; level++){
		biggestRow = (treeCounts[level] > biggestRow) ? treeCounts[level] : biggestRow;
	}


	std::cout << "Biggest row: " << biggestRow  << std::endl;


	const our_size_t stackCapacity = biggestRow;
	const our_size_t blocksPerLaunch = MAX_STACK_ENTRIES / stackCapacity;
	std::cout << "Allowing: " << blocksPerLaunch << " blocks per launch" << std::endl;

	NodeArray<DIM, Float> bfsStackBuffers;
	our_size_t * bfsStackCounters;
	allocDeviceNodeArray(blocksPerLaunch * 2 * stackCapacity, bfsStackBuffers);

	ALLOC_DEBUG_MSG(blocksPerLaunch * 2 * sizeof(our_size_t));
	gpuErrchk( (hipMalloc(&bfsStackCounters, blocksPerLaunch * 2 * sizeof(our_size_t))) );
	
	
	GroupInfoArray<DIM, Float, PPG> cuGroupInfo;
	allocDeviceGroupInfoArray(nGroups, cuGroupInfo);
	copyDeviceGroupInfoArray(nGroups, cuGroupInfo, groupInfo, hipMemcpyHostToDevice);
	
	ParticleArray<DIM, Float> cuParticles;
	allocDeviceParticleArray(n, cuParticles);
	copyDeviceParticleArray(n, cuParticles, particles, hipMemcpyHostToDevice);
	
	InteractionTypeArray(DIM, Float, Mode) cuInteractions;
	allocDeviceVecArray(n, cuInteractions);
	copyDeviceVecArray(n, cuInteractions, interactions, hipMemcpyHostToDevice);
	
	dim3 dimGrid(blocksPerLaunch);
	dim3 dimBlock(threadCt);
	std::cout << "Trying to launch with " << threadCt << " / block with " << blocksPerLaunch << " blocks" << std::endl;
	
	tic;
	traverseTreeKernel<DIM, Float, threadCt, PPG, MAX_LEVELS, INTERACTION_THRESHOLD, Mode, spam><<<dimGrid, dimBlock>>>(nGroups, cuGroupInfo, startDepth, cuTreeLevels, cuTreeCounts, n, cuParticles, cuInteractions, softening, theta, bfsStackCounters, bfsStackBuffers, stackCapacity);
	toc;
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );
	
	copyDeviceVecArray(n, interactions, cuInteractions, hipMemcpyDeviceToHost);
	
	freeDeviceVecArray(cuInteractions);
	freeDeviceParticleArray(cuParticles);
	freeDeviceGroupInfoArray(cuGroupInfo);
	freeDeviceNodeArray(bfsStackBuffers);
	gpuErrchk( (hipFree(bfsStackCounters)) );
	gpuErrchk( (hipFree(cuTreeCounts)) );
	freeDeviceTree<DIM, Float, MAX_LEVELS>(placeHolderLevels);
	gpuErrchk( (hipFree(cuTreeLevels)) );
	
	
	
}


template void traverseTreeCUDA<3, float, 512, 16, 16, 300000, 16, Forces, true>				(our_size_t, GroupInfoArray<3, float, 16>, our_size_t, NodeArray<3, float> *, our_size_t *, our_size_t, ParticleArray<3, float>, InteractionTypeArray(3, float, Forces), float, float, our_size_t);
template void traverseTreeCUDA<3, float, 512, 16, 16, 300000, 16, Forces, false>				(our_size_t, GroupInfoArray<3, float, 16>, our_size_t, NodeArray<3, float> *, our_size_t *, our_size_t, ParticleArray<3, float>, InteractionTypeArray(3, float, Forces), float, float, our_size_t);
template void traverseTreeCUDA<3, float, 512, 16, 16, 300000, 16, CountOnly, false>			(our_size_t, GroupInfoArray<3, float, 16>, our_size_t, NodeArray<3, float> *, our_size_t *, our_size_t, ParticleArray<3, float>, InteractionTypeArray(3, float, CountOnly), float, float, our_size_t);
template void traverseTreeCUDA<3, float, 512, 16, 16, 300000, 16, HashInteractions, false>	(our_size_t, GroupInfoArray<3, float, 16>, our_size_t, NodeArray<3, float> *, our_size_t *, our_size_t, ParticleArray<3, float>, InteractionTypeArray(3, float, HashInteractions), float, float, our_size_t);

